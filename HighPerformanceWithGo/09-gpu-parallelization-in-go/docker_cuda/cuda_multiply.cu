#include "hip/hip_runtime.h"
#include <cmath>
#include <cstdlib>
#include <iostream>

const int ELEMENTS = 1 << 20;

__global__ void multiply(int j, float * a, float * b, float * c) {
  int index = threadIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < j; i += stride)
    c[i] = a[i] * b[i];
}

extern "C" {
  int cuda_multiply(void) {
    float * a, * b, * c;

    hipMallocManaged( & a, ELEMENTS * sizeof(float));
    hipMallocManaged( & b, ELEMENTS * sizeof(float));
    hipMallocManaged( & c, ELEMENTS * sizeof(float));

    for (int i = 0; i < ELEMENTS; i++) {
      a[i] = rand() % 10;
      b[i] = rand() % 10;

    }

    int blockSize = 256;
    int numBlocks = (ELEMENTS + blockSize - 1) / blockSize;
    multiply << < numBlocks, blockSize >>> (ELEMENTS, a, b, c);

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    //for (int k = 0; k < ELEMENTS; k++) {
      //std::cout << k << ":" << a[k] << "*" << b[k] << "=" << c[k] << "\n";
    //}

    hipFree(a);
    hipFree(b);
    hipFree(c);
    return 0;

  }
}
